#include "hip/hip_runtime.h"
#include <chrono>
#include "unsharp_mask.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>     
std::ofstream file;
int blurRadius;
const char *ofilename;
void writeToFile(std::string imagename, int blurRadius, double timeToComplete) // Write to file function for writing time and blur values to .csv
{
	file << imagename << "," << blurRadius << "," << timeToComplete << "\n";
}
int main(int argc, char *argv[])
{
	blurRadius = 5;//Standard blur radius of 5
	const char *ifilename = argc > 1 ? argv[1] : "Images/TESTGPU/GhostTown/ghost-town-8k.ppm";
	ofilename = argc > 2 ? argv[2] : "Images/TestGPU/GhostTown/out-ghost-town-8kBR8.ppm";
	const int blur_radius = argc > 3 ? std::atoi(argv[3]) : blurRadius;

	ppm img; // ppm object
	std::vector<unsigned char> data_in, data_sharp; //Data vectors of in and out image 
	std::string name = "TestDataGPU/GhostTown/GhostTownBR_" + std::to_string(blurRadius)+".csv"; // Not majorly OOP with file name, but does the trick
	file.open(name); // open the file

	std::cout << "Blur Radius..." << blurRadius << std::endl; // Testing, for user knowledge to show current blur radius

	std::cout << "Reading in image..." << std::endl; // Letting the user know that the image file is being read
	img.read(ifilename, data_in);

	std::cout << "Resizing..." << std::endl; // Letting the user know about resizing
	data_sharp.resize(img.w * img.h * img.nchannels);
	std::cout << "Image dimensions: " << img.w << "x" << img.h << std::endl;// Testing, for user knowledge to show image dimensions 

	std::cout << "Starting unsharp mask process..." << std::endl; // Telling the user unsharp process is starting 
	auto t1 = std::chrono::steady_clock::now(); // starting clock
	unsharp_mask(data_sharp.data(), data_in.data(), blur_radius, img.w, img.h, img.nchannels); // calling unsharp function (this handles the whole unsharp process)
	auto t2 = std::chrono::steady_clock::now(); // second clock start
	auto timeTaken = std::chrono::duration<double>(t2 - t1).count(); //subtracting the two times for total time taken.
	std::cout << timeTaken << " seconds.\n"; // Telling user total time taken

	std::cout << "Writing information to file..." << std::endl;
	writeToFile("Lena", blurRadius, timeTaken); // writing information to file for current test

	std::cout << "Writing complete image..." << std::endl;
	img.write(ofilename, data_sharp);// writing image file.

	file.close(); // close file safely.

	return 0;
}

